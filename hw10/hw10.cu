#include "hip/hip_runtime.h"
/* *******************************************************
 * Created By Ethan Kreloff April 4th, 2014.
 * *******************************************************
 * Based off of code from CSCI 5239/4239 Advanced Computer
 * Graphics at the University of Colorado, Boulder.
 * *******************************************************
 * Uses CUDA GPU computing to parallelize a brute force 
 * search.
 * *******************************************************
 * hw10.cu
 * ******************************************************/


/*
 *  CUDA BruteSearch
 */

#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif

/*
 *  Return elapsed wall time since last call (seconds)
 */
static double t0=0;
double Elapsed(void)
{
#ifdef _WIN32
   //  Windows version of wall time
   LARGE_INTEGER tv,freq;
   QueryPerformanceCounter((LARGE_INTEGER*)&tv);
   QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
   double t = tv.QuadPart/(double)freq.QuadPart;
#else
   //  Unix/Linux/OSX version of wall time
   struct timeval tv;
   gettimeofday(&tv,NULL);
   double t = tv.tv_sec+1e-6*tv.tv_usec;
#endif
   double s = t-t0;
   t0 = t;
   return s;
}

/*
 *  Print message to stderr and exit
 */
void Fatal(const char* format , ...)
{
   va_list args;
   va_start(args,format);
   vfprintf(stderr,format,args);
   va_end(args);
   exit(1);
}

/*
 *  Initialize array with random values
 */

void arrayInit(unsigned int x[], const long long int n)
{
    for(unsigned int i = 0; i < n; i++)
    {
        x[i] = i;
    }
}

void shuffle(unsigned int x[], const long long int n)
{
    if (n > 1) 
    {
        for (unsigned int i = 0; i < n - 1; i++)
        {
          size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
          unsigned int t = x[j];
          x[j] = x[i];
          x[i] = t;
        }
    }
}

void RandomInit(unsigned int x[],const long long int n)
{
   arrayInit(x, n);
   shuffle(x, n);
}

/*
 *  Initialize fastest GPU device
 */
int InitGPU(int verbose)
{
   //  Get number of CUDA devices
   int num;
   if (hipGetDeviceCount(&num)) Fatal("Cannot get number of CUDA devices\n");
   if (num<1) Fatal("No CUDA devices found\n");

   //  Get fastest device
   hipDeviceProp_t prop;
   int   MaxDevice = -1;
   int   MaxGflops = -1;
   for (int dev=0;dev<num;dev++)
   {
      if (hipGetDeviceProperties(&prop,dev)) Fatal("Error getting device %d properties\n",dev);
      int Gflops = prop.multiProcessorCount * prop.clockRate;
      if (verbose) printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n",dev,prop.name,1e-6*Gflops,prop.multiProcessorCount,prop.maxThreadsPerBlock);
      if(Gflops > MaxGflops)
      {
         MaxGflops = Gflops;
         MaxDevice = dev;
      }
   }

   //  Print and set device
   if (hipGetDeviceProperties(&prop,MaxDevice)) Fatal("Error getting device %d properties\n",MaxDevice);
   printf("Fastest CUDA Device %d: %s\n",MaxDevice,prop.name);
   hipSetDevice(MaxDevice);

   //  Return max thread count
   return prop.maxThreadsPerBlock;
}

/*
 * Host
 */
long long int BruteSearch(unsigned int x[], const long long int match, const long long int n)
{   
   int index = -1;

   for (unsigned int i=0; i<n; i++){
        if(x[i] == match) index = i;
   }
   
   return index;

}

/*
 *  Subset Device Function
 */

__global__ void findInSubset(int *index, const unsigned int x[], const unsigned int n, const unsigned int match)
{
   unsigned int j = blockIdx.x*blockDim.x+threadIdx.x;
   unsigned int i = blockIdx.y*blockDim.y+threadIdx.y;
   if(x[i*j] == match) *index = i*j;
}

/*
 *  Device Function
 */
 
 void ParallelBrute(int *index, const unsigned int x[], const unsigned int n, const unsigned int match)
{
   //  Calculate matrix dimensions
   int N = n*sizeof(unsigned int);

   // Allocate device memory
   unsigned int* list;
   int *position;
   if (hipMalloc((void**)&list,N)) Fatal("Cannot allocate device memory list\n");
   if (hipMalloc((void**)&position,sizeof(int))) Fatal("Cannot allocate device memory position\n");

   // Copy A and B from host to device
   if (hipMemcpy(list,x,N,hipMemcpyHostToDevice)) Fatal("Cannot copy list from host to device\n");
   if (hipMemcpy(position,index,sizeof(int),hipMemcpyHostToDevice)) Fatal("Cannot copy position from host to device\n");

   // Set size of block to Bw x Bw, and Bn x Bn blocks
   dim3 threads(10,10);
   dim3 grid((n/2)/threads.x,(n/2)/threads.y);

   // Execute the kernel
   findInSubset<<<grid,threads>>>(position,list,n,match);
   if (hipGetLastError()) Fatal("findinsubset failed\n");

   // Copy C from device to host
   if (hipMemcpy(index,position,sizeof(int),hipMemcpyDeviceToHost)) Fatal("Cannot copy position from device to host\n");

   //  Free device memory
   hipFree(list);
   hipFree(position);

}


/*
 *  main
 */
int main(int argc, char* argv[])
{

   //  Process options
   int opt;
   int verbose=0;
   while ((opt=getopt(argc,argv,"v"))!=-1)
   {
      if (opt=='v')
         verbose++;
      else
         Fatal("Usage: [-v] <number to search for> <length of list>\n");
   }
   argc -= optind;
   argv += optind;

   
   //  Get number to search for and length of list
   if (argc!=2) Fatal("Usage: [-v] <number to search for> <length of list>\n");
   long long int find = atoi(argv[0]);
   if (find<0) Fatal("Number must not be negative.");
   long long int n = atoi(argv[1]);
   if (n<1) Fatal("Length of list must be positive.");
   printf("Looking for %lld in a list %lld items long.\n", find, n);
   

   //  Initialize GPU
   int Mw = InitGPU(verbose);
   if (Mw<n) Fatal("Thread count %lld exceeds threads per block of %d\n", n, Mw);
   
   // Allocate host memory
   unsigned int* array = (unsigned int*)malloc(n);
   if (!array) Fatal("Cannot allocate host memory\n");
   
   // Initialize List
   srand(time(NULL));
   RandomInit(array, n);
   
   Elapsed();
   long long int hostIndex = BruteSearch(array, find, n);
   double Th = Elapsed();
   
   //  Free host memory
   free(array);

   if(hostIndex < 0){
        printf("Host didn't find %lld in %6.3f s\n", find, Th);
   }else{
        printf("Host found %lld at position %lld in %6.22f s\n", find, hostIndex + 1, Th);
   }

   //  Done
   return 0;
}
